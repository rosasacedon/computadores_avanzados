
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 500

__global__ void VecAdd(int* DA, int* DB, int* DC)
{
	int i = threadIdx.x;
    DC[i] = DA[i] + DB[i];//se sumara los componentes de un vector y de otro
}

int main()
{ 
  int HA[N], HB[N], HC[N];
  int *DA, *DB, *DC;//se tienen que generar de manera dinamica
  int i; int size = N*sizeof(int); //es el tamaño de reserva de espacio
  
  hipFree(0);

  // reservamos espacio en la memoria global del device
  hipMalloc((void**)&DA, size);
  hipMalloc((void**)&DB, size);
  hipMalloc((void**)&DC, size);
  
  // inicializamos HA y HB
  for (i=0; i<N; i++) {HA[i]=-i; HB[i] = 3*i;}
  
  // copiamos HA y HB del host a DA y DB en el device, respectivamente
  hipMemcpy(DA, HA, size, hipMemcpyHostToDevice);
  hipMemcpy(DB, HB, size, hipMemcpyHostToDevice);
  
  // llamamos al kernel (1 bloque de N hilos)
  VecAdd <<<1, N>>>(DA, DB, DC);	// N hilos ejecutan el kernel en paralelo, los hilos al ser en una dimension se identifican por thread.x
  
  // copiamos el resultado, que está en la memoria global del device, (DC) al host (a HC)
  hipMemcpy(HC, DC, size, hipMemcpyDeviceToHost);
  
  // liberamos la memoria reservada en el device
  hipFree(DA); hipFree(DB); hipFree(DC);  
  
  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  // esta comprobación debe quitarse una vez que el programa es correcto (p. ej., para medir el tiempo de ejecución)
  for (i = 0; i < N; i++) // printf("%d + %d = %d\n",HA[i],HB[i],HC[i]);
    if (HC[i]!= (HA[i]+HB[i])) 
      {printf("error en componente %d\n", i); break;}
    
  return 0;
} 
