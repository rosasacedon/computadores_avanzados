
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 100000
#define tb 512	// tamaño bloque

__global__ void VecAdd(int* DA, int* DB, int* DC)
{
	int ii = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i=ii; i<N; i+=stride)
	    DC[i] = DA[i] + DB[i];
}

hipError_t testCuErr(hipError_t result)
{
  if (result != hipSuccess) {
    printf("CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);	// si no se cumple, se aborta el programa
  }
  return result;
}

int main()
{ hipFree(0);
  int *HA, *HB, *HC, *DA, *DB, *DC;
  int i, dg; int size = N*sizeof(int);

  // HA = (int*)malloc(size); HB = (int*)malloc(size); HC = (int*)malloc(size);
  
  // reservamos espacio en la memoria global del device
  testCuErr(hipMalloc((void**)&DA, size));
  testCuErr(hipMalloc((void**)&DB, size));
  testCuErr(hipMalloc((void**)&DC, size));

  // reservamos espacio en la memoria global del host
  testCuErr(hipHostMalloc((void**)&HA, size, hipHostMallocDefault));
  testCuErr(hipHostMalloc((void**)&HB, size, hipHostMallocDefault));
  testCuErr(hipHostMalloc((void**)&HC, size, hipHostMallocDefault));
     
  // inicializamos HA y HB
  for (i=0; i<N; i++) {HA[i]=-i; HB[i] = 3*i;}
  
  // copiamos HA y HB del host a DA y DB en el device, respectivamente
  testCuErr(hipMemcpy(DA, HA, size, hipMemcpyHostToDevice));
  testCuErr(hipMemcpy(DB, HB, size, hipMemcpyHostToDevice));
      
  dg = (N+tb-1)/tb; if (dg>65535) dg=65535;
  // llamamos al kernel
  VecAdd <<<dg, tb>>>(DA, DB, DC);	// N o más hilos ejecutan el kernel en paralelo
  testCuErr(hipGetLastError());
  
  // copiamos el resultado, que está en la memoria global del device, (DC) al host (a HC)
  testCuErr(hipMemcpy(HC, DC, size, hipMemcpyDeviceToHost));
    
  // liberamos la memoria reservada en el device
  testCuErr(hipFree(DA)); testCuErr(hipFree(DB)); testCuErr(hipFree(DC));  
    
  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  for (i = 0; i < N; i++) // printf("%d + %d = %d\n",HA[i],HB[i],HC[i]);
    if (HC[i]!= (HA[i]+HB[i])) 
		{printf("error en componente %d\n", i); break;}
 
  // free(HA); free(HB); free(HC);
  hipHostFree(HA); hipHostFree(HB); hipHostFree(HC);
  return 0;
} 
