
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 600

__global__ void VecAdd(int* DA, int* DB, int* DC)
{
	int i=0;
	for (i=0;i<N;i++){
		DC[i] = DA[i] + DB[i];
  }
}

int main()
{ 
  int HA[N], HB[N], HC[N];
  int *DA, *DB, *DC;//se tienen que generar de manera dinamica
  int i; int size = N*sizeof(int); //es el tamaño de reserva de espacio
  
  hipFree(0);
  hipError_t tester;

  // reservamos espacio en la memoria global del device
  tester=hipMalloc((void**)&DA, size);
  if(tester!=hipSuccess){
    printf("Error en cuda %s", hipGetErrorString(tester));
    exit(0);
  }
  tester=hipMalloc((void**)&DB, size);
  if(tester!=hipSuccess){
    printf("Error en cuda %s", hipGetErrorString(tester));
    exit(0);
  }
  tester=hipMalloc((void**)&DC, size);
  if(tester!=hipSuccess){
    printf("Error en cuda %s", hipGetErrorString(tester));
    exit(0);
  }
  
  // inicializamos HA y HB
  for (i=0; i<N; i++) {HA[i]=-i; HB[i] = 3*i;}
  
  // copiamos HA y HB del host a DA y DB en el device, respectivamente
  tester=hipMemcpy(DA, HA, size, hipMemcpyHostToDevice);
  if(tester!=hipSuccess){
    printf("Error en cuda %s", hipGetErrorString(tester));
    exit(0);
  }
  tester=hipMemcpy(DB, HB, size, hipMemcpyHostToDevice);
  if(tester!=hipSuccess){
    printf("Error en cuda %s", hipGetErrorString(tester));
    exit(0);
  }
  
  // llamamos al kernel (1 bloque de N hilos)
  
  VecAdd <<<1, 1>>>(DA, DB, DC);	// N hilos ejecutan el kernel en paralelo, los hilos al ser en una dimension se identifican por thread.x
  tester = hipGetLastError();
 if(tester!=hipSuccess){
    printf("Error en cuda %s", hipGetErrorString(tester));
    exit(0);
  }
  
  // copiamos el resultado, que está en la memoria global del device, (DC) al host (a HC)
  tester=hipMemcpy(HC, DC, size, hipMemcpyDeviceToHost);
   if(tester!=hipSuccess){
    printf("Error en cuda %s", hipGetErrorString(tester));
    exit(0);
  }
  // liberamos la memoria reservada en el device
  tester=hipFree(DA); hipFree(DB); hipFree(DC);  
   if(tester!=hipSuccess){
    printf("Error en cuda %s", hipGetErrorString(tester));
    exit(0);
  }
  
  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  // esta comprobación debe quitarse una vez que el programa es correcto (p. ej., para medir el tiempo de ejecución)
  for (i = 0; i < N; i++) // printf("%d + %d = %d\n",HA[i],HB[i],HC[i]);
    if (HC[i]!= (HA[i]+HB[i])) 
       {printf("error en componente %d\n", i); break;}
    
  return 0;
} 
